#include "hip/hip_runtime.h"
#include "Preprocessing.h"
static void HandleError( hipError_t err, const char *file,  int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),  file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void cuda_normalization(float *data, int rows, int columns) {
    int total_threads_count = blockDim.x * gridDim.x;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int min, max;
    float max_float = FLT_MAX;

    for (int i = tid+1; i < columns; i=i+total_threads_count) {
        min = max_float; max = 0;
        for (int j = 0; j < rows; ++j) {
            if (*(data + (j*columns)+i) < min) {
                min = *(data + (j*columns)+i);
            } else if (*(data + (j*columns)+i) > max) {
                max = *(data + (j*columns)+i);
            }
        }

        float max_min_reciprocal = max - min;
        if (max_min_reciprocal == 0) {
            continue;
        }
        max_min_reciprocal = 1. / max_min_reciprocal;

        for (int j = 0; j < rows; ++j) {
            *(data + (j*columns)+i) = (*(data + (j*columns)+i) - min) * max_min_reciprocal;
        }
    }
}

__global__ void cuda_standarization(float *data, int rows, int columns) {
    int total_threads_count = blockDim.x * gridDim.x;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float var, ave, amo;

    for (int i = tid+1; i < columns; i=i+total_threads_count) {
        amo = 0, var = 0;
        for (int j = 0; j < rows; ++j) {
            amo = amo + *(data + (j * columns) + i);
        }
        ave  = amo / float(rows);

        for (int j = 0; j < rows; ++j) {
            float factor = *(data + (j * columns) + i) - ave;
            var = var + (factor * factor);
        }

        if (var == 0) {
            for (int j = 0; j < rows; j++) {
                *(data + (j * columns) + i) = *(data + (j * columns) + i) / 255.;
            }
            continue;
        }

        float sd_reciprocal = 1./sqrt(var);

        for (int j = 0; j < rows; j++) {
            *(data + (j * columns) + i) = (*(data + (j * columns) + i) - ave) * sd_reciprocal;
        }
    }
}

Preprocessing::Preprocessing() {
}


Preprocessing::~Preprocessing() {
}

void Preprocessing::Normalization(float *data, int rows, int columns,
        int threads_count_per_block, int blocks_count) {
    hipDeviceProp_t cuda_properties; // information about gpu
    HANDLE_ERROR(hipGetDeviceProperties( &cuda_properties, 0));

    // copy data to compute from RAM into gpu device memory
    float *cuda_data;
    int data_size = sizeof(float) * rows * columns;
    HANDLE_ERROR(hipMalloc((void**)&cuda_data, data_size));
    HANDLE_ERROR(hipMemcpy(cuda_data, data, data_size, hipMemcpyHostToDevice));

    // measure time using cuda events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // standarize
    cuda_normalization<<<blocks_count, threads_count_per_block>>>(cuda_data, rows, columns);
    hipEventRecord(stop);

    // copy computed data from gpu device memory to host RAM
    HANDLE_ERROR(hipMemcpy(data, cuda_data, data_size, hipMemcpyDeviceToHost));

    // print elapsed time
    hipEventSynchronize(stop);
    float elapsed_time = 0;
    hipEventElapsedTime(&elapsed_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("Czas obliczen normalizacja: %f\n", elapsed_time/1000);
}


void Preprocessing::Standarization(float *data, int rows, int columns,
        int threads_count_per_block, int blocks_count) {
    hipDeviceProp_t cuda_properties; // information about gpu
    HANDLE_ERROR(hipGetDeviceProperties( &cuda_properties, 0));

    // copy data to compute into gpu device memory
    float *cuda_data;
    int data_size = sizeof(float) * rows * columns;
    HANDLE_ERROR(hipMalloc((void**)&cuda_data, data_size));
    HANDLE_ERROR(hipMemcpy(cuda_data, data, data_size, hipMemcpyHostToDevice));

    // measure time using cuda events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // standarize
    cuda_standarization<<<blocks_count, threads_count_per_block>>>(cuda_data, rows, columns);
    hipEventRecord(stop);

    // copy computed data from gpu device memory to host RAM
    HANDLE_ERROR(hipMemcpy(data, cuda_data, data_size, hipMemcpyDeviceToHost));

    // print elapsed time
    hipEventSynchronize(stop);
    float elapsed_time = 0;
    hipEventElapsedTime(&elapsed_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("Czas obliczen standaryzacja: %f\n", elapsed_time/1000);
}