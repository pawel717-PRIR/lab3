#include "hip/hip_runtime.h"
#include "KnnAlgorithm.h"

KnnAlgorithm::KnnAlgorithm() {
    //ctor
}

KnnAlgorithm::~KnnAlgorithm() {
    //dtor
}

void KnnAlgorithm::fit(Data * data, int percent) {
    this->train_rows = (data->rows * percent) / 100;
    this->columns = data->columns;
    this->test_rows = data->rows - train_rows;
    this->train_data = data->data;
    this->test_data = data->data + (columns * train_rows);
}

float KnnAlgorithm::predict() {
    int closest_neighbour_index, accurate_predictions = 0;
    float max_float = std::numeric_limits<float>::max();

    for (int current_test_row=0; current_test_row < test_rows; ++current_test_row) {
        float closest_neighbour_distance = max_float;
        float* tst = test_data + (columns * current_test_row);
        // for each row in train dataset
        for (int i = 0; i < train_rows; ++i) {
            float* tr = train_data + (i * columns) + 1;
            // calculate eucidlean metric and get the closest one
            float sum = 0;
            for(int j = 1; j < columns; ++j, ++tr) {
                float difference = *(tr) - *(tst +j);
                sum = sum + (difference * difference);
            }
            // distance is euclidean metric for current_test_row and i-th train data
            // if our data is closer to that row from train data update closest_neighbour_distance and and closest_neighbour_index
            if(sum < closest_neighbour_distance) {
                closest_neighbour_distance = sum;
                closest_neighbour_index = i;
            }
        }
        // now we have found closest neighbour and have index of that neighbour in closest_neighbour_index variable
        // so let's get target class of that neighbour (predicted class) and check if the prediction is accurate
        if(*(test_data + (current_test_row * columns)) == *(train_data + (closest_neighbour_index * columns))) {
            // if prediction is accurate increment accurate predictions counter
            accurate_predictions = accurate_predictions + 1;
        }
    }

    //printf("Czas obliczen knn: %f\n", MPI_Wtime() - startTime);

    return (accurate_predictions / float(test_rows)) * 100;
}
