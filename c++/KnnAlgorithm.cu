#include "hip/hip_runtime.h"
#include "KnnAlgorithm.h"
static void HandleError( hipError_t err, const char *file,  int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),  file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
//__device__ int * accurate_predictions = 0;
__global__ void cuda_knn_predict(float *data, int train_rows, int test_rows, int columns, int * accurate_predictions) {
    int total_threads_count = blockDim.x * gridDim.x;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int closest_neighbour_index;
    float max_float = FLT_MAX;
    float* train_data = data;
    float* test_data = data + (columns * train_rows);

    for (int current_test_row=tid; current_test_row < test_rows; current_test_row=current_test_row+total_threads_count) {
        float closest_neighbour_distance = max_float;
        float* tst = test_data + (columns * current_test_row);
        // for each row in train dataset
        for (int i = 0; i < train_rows; ++i) {
            float* tr = train_data + (i * columns) + 1;
            // calculate eucidlean metric and get the closest one
            float sum = 0;
            for (int j = 1; j < columns; ++j, ++tr) {
                float difference = *(tr) - *(tst +j);
                sum = sum + (difference * difference);
            }
            // distance is euclidean metric for current_test_row and i-th train data
            // if our data is closer to that row from train data update closest_neighbour_distance and and closest_neighbour_index
            if(sum < closest_neighbour_distance) {
                closest_neighbour_distance = sum;
                closest_neighbour_index = i;
            }
        }
        // now we have found closest neighbour and have index of that neighbour in closest_neighbour_index variable
        // so let's get target class of that neighbour (predicted class) and check if the prediction is accurate
        if(*(test_data + (current_test_row * columns)) == *(train_data + (closest_neighbour_index * columns))) {
            // if prediction is accurate increment accurate predictions counter
            //atomicAdd(accurate_predictions, 1);
            accurate_predictions[tid] = 1;
        } else {
            accurate_predictions[tid] = 0;
        }
    }
}

KnnAlgorithm::KnnAlgorithm() {
    //ctor
}

KnnAlgorithm::~KnnAlgorithm() {
    //dtor
}

void KnnAlgorithm::fit(Data * data, int percent) {
    this->train_rows = (data->rows * percent) / 100;
    this->columns = data->columns;
    this->test_rows = data->rows - train_rows;
    this->train_data = data->data;
    this->test_data = data->data + (columns * train_rows);
}

float KnnAlgorithm::predict() {
    int* accurate_predictions;
    int* cuda_accurate_predictions;


    hipDeviceProp_t cuda_properties; // information about gpu
    HANDLE_ERROR(hipGetDeviceProperties( &cuda_properties, 0));
    int threads_count_per_block = cuda_properties.maxThreadsPerBlock; // use as many threads as possible on this device
    if(threads_count_per_block > this->test_rows) {
        threads_count_per_block = this->test_rows;
    }
    int blocks_count = (this->test_rows + threads_count_per_block - 1) / threads_count_per_block;
    int max_blocks_count = cuda_properties.maxGridSize[0];
    if(blocks_count > max_blocks_count) {
        blocks_count = max_blocks_count;
    }

    // copy data to compute into gpu device memory
    float *cuda_data;
    int data_size = sizeof(float) * (this->test_rows + this->train_rows) * this->columns;
    HANDLE_ERROR(hipMalloc((void**)&cuda_data, data_size));
    HANDLE_ERROR(hipMemcpy(cuda_data, this->train_data, data_size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMalloc((void**)&cuda_accurate_predictions, test_rows * sizeof(int)));

    // measure time using cuda events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // perform knn prediction
    cuda_knn_predict<<<blocks_count, threads_count_per_block>>>(cuda_data, this->train_rows, this->test_rows, this->columns, cuda_accurate_predictions);
    hipEventRecord(stop);

    // print elapsed time
    hipEventSynchronize(stop);
    float elapsed_time = 0;
    hipEventElapsedTime(&elapsed_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("Czas obliczen knn: %f\n", elapsed_time/1000);

    // copy from gpu device memory to host RAM
    //int* cuda_acc = NULL;
   // HANDLE_ERROR(hipMemcpyFromSymbol((void**)&cuda_acc, "accurate_predictions", sizeof(cuda_acc), 0, hipMemcpyDeviceToHost));
    accurate_predictions = (int*) malloc (test_rows * sizeof(int));
    HANDLE_ERROR(hipMemcpy(accurate_predictions, cuda_accurate_predictions, sizeof(int) * test_rows,
            hipMemcpyDeviceToHost));
    int sum = 0;
    for (int i=0; i<test_rows; i++) {
        sum += *(accurate_predictions+i);
    }

    return (sum / float(test_rows)) * 100;
}
